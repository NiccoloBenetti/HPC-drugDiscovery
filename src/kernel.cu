#include <hip/hip_runtime.h>
#include <cmath>

// Kernel CUDA bidimensionale per calcolare le distanze tra atomi
 __global__ void calculateDistancesKernel2D(float* posA_x, float* posA_y, float* posA_z,
                                           float* posB_x, float* posB_y, float* posB_z,
                                           float* distances, int numA, int numB) {
    // Calcola gli indici bidimensionali del thread all'interno della griglia
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Indice per posA (molecola A)
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // Indice per posB (molecola B)

    // Verifica che gli indici siano validi
    if (i < numA && j < numB) {
        // Calcolo della distanza euclidea tra l'atomo i di molA e l'atomo j di molB
        float dx = posA_x[i] - posB_x[j];
        float dy = posA_y[i] - posB_y[j];
        float dz = posA_z[i] - posB_z[j];
        distances[i * numB + j] = sqrtf(dx * dx + dy * dy + dz * dz);
    }
}

// Funzione wrapper per chiamare il kernel CUDA bidimensionale
extern "C" void launchDistanceKernel2D(float* d_posA_x, float* d_posA_y, float* d_posA_z,
                                       float* d_posB_x, float* d_posB_y, float* d_posB_z,
                                       float* d_distances, int numA, int numB, int blockSizeX, int blockSizeY) {
    // Definisci la dimensione del blocco e della griglia
    dim3 threadsPerBlock(blockSizeX, blockSizeY);  // Blocchi 2D di thread
    dim3 blocksPerGrid((numA + blockSizeX - 1) / blockSizeX, 
                       (numB + blockSizeY - 1) / blockSizeY);  // Griglia 2D di blocchi

    // Lancia il kernel CUDA bidimensionale
    calculateDistancesKernel2D<<<blocksPerGrid, threadsPerBlock>>>(d_posA_x, d_posA_y, d_posA_z,
                                                                   d_posB_x, d_posB_y, d_posB_z,
                                                                   d_distances, numA, numB);
}

__global__ void calculateHydrogenBondKernel(float* donor_x, float* donor_y, float* donor_z,
                                            float* hydrogen_x, float* hydrogen_y, float* hydrogen_z,
                                            float* acceptor_x, float* acceptor_y, float* acceptor_z,
                                            float* distances, float* angles, int numDonors, int numAcceptors) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Indice per il donatore
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // Indice per l'accettore

    if (i < numDonors && j < numAcceptors) {
        // Calcolo della distanza euclidea tra donatore e accettore
        float dx = donor_x[i] - acceptor_x[j];
        float dy = donor_y[i] - acceptor_y[j];
        float dz = donor_z[i] - acceptor_z[j];
        distances[i * numAcceptors + j] = sqrtf(dx * dx + dy * dy + dz * dz);

        // Calcolo dell'angolo tra donatore, idrogeno e accettore
        float hx = hydrogen_x[i], hy = hydrogen_y[i], hz = hydrogen_z[i];
        float dhx = donor_x[i] - hx, dhy = donor_y[i] - hy, dhz = donor_z[i] - hz;
        float ahx = acceptor_x[j] - hx, ahy = acceptor_y[j] - hy, ahz = acceptor_z[j] - hz;

        float dotProduct = dhx * ahx + dhy * ahy + dhz * ahz;
        float mag_dh = sqrtf(dhx * dhx + dhy * dhy + dhz * dhz);
        float mag_ah = sqrtf(ahx * ahx + ahy * ahy + ahz * ahz);
        angles[i * numAcceptors + j] = acosf(dotProduct / (mag_dh * mag_ah)) * 180.0f / M_PI;
    }
}

    // Funzione wrapper per chiamare il kernel CUDA per il calcolo dei legami a idrogeno
    extern "C" void launchHydrogenBondKernel(float* d_donor_x, float* d_donor_y, float* d_donor_z,
                                         float* d_hydrogen_x, float* d_hydrogen_y, float* d_hydrogen_z,
                                         float* d_acceptor_x, float* d_acceptor_y, float* d_acceptor_z,
                                         float* d_distances, float* d_angles,
                                         int numDonors, int numAcceptors, int blockSizeX, int blockSizeY) {
    // Definisci la dimensione del blocco e della griglia
    dim3 threadsPerBlock(blockSizeX, blockSizeY);  // Blocchi 2D di thread
    dim3 blocksPerGrid((numDonors + blockSizeX - 1) / blockSizeX, 
                       (numAcceptors + blockSizeY - 1) / blockSizeY);  // Griglia 2D di blocchi

    // Lancia il kernel CUDA bidimensionale per il calcolo dei legami a idrogeno
    calculateHydrogenBondKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_donor_x, d_donor_y, d_donor_z,
        d_hydrogen_x, d_hydrogen_y, d_hydrogen_z,
        d_acceptor_x, d_acceptor_y, d_acceptor_z,
        d_distances, d_angles,
        numDonors, numAcceptors);
}

__global__ void calculateHalogenBondKernel(float* donor_x, float* donor_y, float* donor_z,
                                           float* halogen_x, float* halogen_y, float* halogen_z,
                                           float* acceptor_x, float* acceptor_y, float* acceptor_z,
                                           float* any_x, float* any_y, float* any_z,
                                           float* distances, float* firstAngles, float* secondAngles,
                                           int numDonors, int numAcceptors, float maxDistance,
                                           float minAngle1, float maxAngle1, float minAngle2, float maxAngle2) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Indice per i donatori
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // Indice per gli accettori

    if (i < numDonors && j < numAcceptors) {
        // Calcolo della distanza euclidea tra il donatore e l'accettore
        float dx = donor_x[i] - acceptor_x[j];
        float dy = donor_y[i] - acceptor_y[j];
        float dz = donor_z[i] - acceptor_z[j];
        float distance = sqrtf(dx * dx + dy * dy + dz * dz);

        // Calcolo degli angoli
        float hx = halogen_x[i], hy = halogen_y[i], hz = halogen_z[i];
        float ax = acceptor_x[j], ay = acceptor_y[j], az = acceptor_z[j];
        float anyx = any_x[j], anyy = any_y[j], anyz = any_z[j];

        // Primo angolo: tra donatore, alogeno e accettore
        float dhx = donor_x[i] - hx, dhy = donor_y[i] - hy, dhz = donor_z[i] - hz;
        float ahx = ax - hx, ahy = ay - hy, ahz = az - hz;
        float dotProduct1 = dhx * ahx + dhy * ahy + dhz * ahz;
        float mag_dh = sqrtf(dhx * dhx + dhy * dhy + dhz * dhz);
        float mag_ah = sqrtf(ahx * ahx + ahy * ahy + ahz * ahz);
        float firstAngle = acosf(dotProduct1 / (mag_dh * mag_ah)) * 180.0f / M_PI;

        // Secondo angolo: tra accettore, alogeno e "any"
        float ahhx = ax - hx, ahhy = ay - hy, ahhz = az - hz;
        float aax = anyx - ax, aay = anyy - ay, aaz = anyz - az;
        float dotProduct2 = ahhx * aax + ahhy * aay + ahhz * aaz;
        float mag_ahh = sqrtf(ahhx * ahhx + ahhy * ahhy + ahhz * ahhz);
        float mag_aa = sqrtf(aax * aax + aay * aay + aaz * aaz);
        float secondAngle = acosf(dotProduct2 / (mag_ahh * mag_aa)) * 180.0f / M_PI;

        // Salva le distanze e gli angoli solo se soddisfano i criteri
        if (distance <= maxDistance && firstAngle >= minAngle1 && firstAngle <= maxAngle1 && 
            secondAngle >= minAngle2 && secondAngle <= maxAngle2) {
            distances[i * numAcceptors + j] = distance;
            firstAngles[i * numAcceptors + j] = firstAngle;
            secondAngles[i * numAcceptors + j] = secondAngle;
        } else {
            distances[i * numAcceptors + j] = -1.0f;  // Usa un valore negativo per indicare nessuna interazione
        }
    }
}


extern "C" void launchHalogenBondKernel(float* d_donor_x, float* d_donor_y, float* d_donor_z,
                                        float* d_halogen_x, float* d_halogen_y, float* d_halogen_z,
                                        float* d_acceptor_x, float* d_acceptor_y, float* d_acceptor_z,
                                        float* d_any_x, float* d_any_y, float* d_any_z,
                                        float* d_distances, float* d_firstAngles, float* d_secondAngles,
                                        int numDonors, int numAcceptors, int blockSizeX, int blockSizeY,
                                        float maxDistance, float minAngle1, float maxAngle1,
                                        float minAngle2, float maxAngle2) {
    // Definisci la dimensione dei blocchi e della griglia
    dim3 threadsPerBlock(blockSizeX, blockSizeY);
    dim3 blocksPerGrid((numDonors + blockSizeX - 1) / blockSizeX, 
                       (numAcceptors + blockSizeY - 1) / blockSizeY);

    // Lancia il kernel per il calcolo dei legami di alogeni
    calculateHalogenBondKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_donor_x, d_donor_y, d_donor_z,
        d_halogen_x, d_halogen_y, d_halogen_z,
        d_acceptor_x, d_acceptor_y, d_acceptor_z,
        d_any_x, d_any_y, d_any_z,
        d_distances, d_firstAngles, d_secondAngles,
        numDonors, numAcceptors, maxDistance, minAngle1, maxAngle1, minAngle2, maxAngle2);
}

__global__ void calculateCationAnionKernel(float* cation_x, float* cation_y, float* cation_z,
                                           float* anion_x, float* anion_y, float* anion_z,
                                           float* distances, int numCations, int numAnions, float maxDistance) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Indice per i cationi
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // Indice per gli anioni

    if (i < numCations && j < numAnions) {
        // Calcolo della distanza tra catione e anione
        float dx = cation_x[i] - anion_x[j];
        float dy = cation_y[i] - anion_y[j];
        float dz = cation_z[i] - anion_z[j];
        float distance = sqrtf(dx * dx + dy * dy + dz * dz);

        // Verifica se la distanza è inferiore alla distanza massima per l'interazione ionica
        if (distance <= maxDistance) {
            distances[i * numAnions + j] = distance;
        } else {
            distances[i * numAnions + j] = -1.0f;  // Nessuna interazione
        }
    }
}

extern "C" void launchIonicInteractionsKernel_CationAnion(float* d_cation_x, float* d_cation_y, float* d_cation_z,
                                                          float* d_anion_x, float* d_anion_y, float* d_anion_z,
                                                          float* d_distances, int numCations, int numAnions, 
                                                          int blockSizeX, int blockSizeY, float maxDistance) {
    dim3 threadsPerBlock(blockSizeX, blockSizeY);
    dim3 blocksPerGrid((numCations + blockSizeX - 1) / blockSizeX, 
                       (numAnions + blockSizeY - 1) / blockSizeY);

    // Lancia il kernel per Cationi-Anioni
    calculateCationAnionKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_cation_x, d_cation_y, d_cation_z,
        d_anion_x, d_anion_y, d_anion_z,
        d_distances, numCations, numAnions, maxDistance);
}

__global__ void calculateCationRingKernel(float* cation_x, float* cation_y, float* cation_z,
                                          float* ring_centroid_x, float* ring_centroid_y, float* ring_centroid_z,
                                          float* ring_normal_x, float* ring_normal_y, float* ring_normal_z,
                                          float* distances, float* angles, int numCations, int numRings, 
                                          float maxDistance, float minAngle, float maxAngle) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Indice per i cationi
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // Indice per gli anelli aromatici

    if (i < numCations && j < numRings) {
        // Calcolo della distanza tra catione e il centro dell'anello aromatico
        float dx = cation_x[i] - ring_centroid_x[j];
        float dy = cation_y[i] - ring_centroid_y[j];
        float dz = cation_z[i] - ring_centroid_z[j];
        float distance = sqrtf(dx * dx + dy * dy + dz * dz);

        // Verifica se la distanza è inferiore alla distanza massima
        if (distance <= maxDistance) {
            // Calcolo dell'angolo tra il catione e il vettore normale all'anello
            float dotProduct = dx * ring_normal_x[j] + dy * ring_normal_y[j] + dz * ring_normal_z[j];
            float magnitude_cation = sqrtf(dx * dx + dy * dy + dz * dz);
            float magnitude_normal = sqrtf(ring_normal_x[j] * ring_normal_x[j] +
                                           ring_normal_y[j] * ring_normal_y[j] +
                                           ring_normal_z[j] * ring_normal_z[j]);
            float angle = acosf(dotProduct / (magnitude_cation * magnitude_normal)) * 180.0f / M_PI;

            // Verifica se l'angolo è compreso nell'intervallo richiesto
            if (angle >= minAngle && angle <= maxAngle) {
                distances[i * numRings + j] = distance;
                angles[i * numRings + j] = angle;
            } else {
                distances[i * numRings + j] = -1.0f;  // Nessuna interazione
            }
        } else {
            distances[i * numRings + j] = -1.0f;  // Nessuna interazione
        }
    }
}

extern "C" void launchIonicInteractionsKernel_CationRing(float* d_cation_x, float* d_cation_y, float* d_cation_z,
                                                         float* d_ring_centroid_x, float* d_ring_centroid_y, float* d_ring_centroid_z,
                                                         float* d_ring_normal_x, float* d_ring_normal_y, float* d_ring_normal_z,
                                                         float* d_distances, float* d_angles, int numCations, int numRings, 
                                                         int blockSizeX, int blockSizeY, float maxDistance, float minAngle, float maxAngle) {
    dim3 threadsPerBlock(blockSizeX, blockSizeY);
    dim3 blocksPerGrid((numCations + blockSizeX - 1) / blockSizeX, 
                       (numRings + blockSizeY - 1) / blockSizeY);

    // Lancia il kernel per Cationi-Anelli Aromatici
    calculateCationRingKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_cation_x, d_cation_y, d_cation_z,
        d_ring_centroid_x, d_ring_centroid_y, d_ring_centroid_z,
        d_ring_normal_x, d_ring_normal_y, d_ring_normal_z,
        d_distances, d_angles, numCations, numRings, maxDistance, minAngle, maxAngle);
}






